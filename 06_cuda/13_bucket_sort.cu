
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucket_init(int *bucket) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    bucket[i] = 0;
}

__global__ void bucket_add(int *key, int *bucket) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&bucket[key[i]], 1);
}

__global__ void bucket_scan( int *bucket,int *a, int range) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    // prefix sum     
    for(int j=1; j<range; j<<=1){
        a[i] = bucket[i];
        __syncthreads();
        bucket[i] += a[i-j];
        __syncthreads();
    }
}

__global__ void bucket_assignment(int *key, int *bucket, int n, int range){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(int j=0; j<range; j++){
        if(i < bucket[j] && i >= bucket[j-1]){
            key[i] = j;
            return;
        }
    }
}

int main() {
    int n = 50;
    int range = 5;
    int *key;
    hipMallocManaged(&key, n*sizeof(int));

    for (int i=0; i<n; i++) {
        key[i] = rand() % range;
        printf("%d ",key[i]);
    }
    printf("\n");

    int *bucket, *a;
    hipMallocManaged(&bucket, range*sizeof(int));
    hipMallocManaged(&a, range*sizeof(int));

    bucket_init<<<1,range>>>(bucket);
    hipDeviceSynchronize();

    bucket_add<<<1,n>>>(key, bucket);
    hipDeviceSynchronize();

    bucket_scan<<<1,range>>>(bucket ,a, range);
    hipDeviceSynchronize();

    bucket_assignment<<<1,n>>>(key, bucket, n, range);
    hipDeviceSynchronize();

    for (int i=0; i<n; i++) {
        printf("%d ",key[i]);
    }
    printf("\n");

    hipFree(key);
    hipFree(bucket);
}
